#include "hip/hip_runtime.h"
/**
 * GNMT LSTM GEMM Optimization.
 * Part of the code comes from `https://github.com/tbennun/cudnn-training`.
 *
 * Author: Wil Kong
 * Date: 01/22/2018 Mon
 */

#include "common.cuh"

//////////////////////////////////////////////////////////////////////////////

int main(int argc, char *argv[])  {
    uint64_t seq_length = 38, input_size = 1024, hidden_size = 1024, batch_size = 32;
    uint64_t num_layer = 3;
    uint64_t num_mat = 4;  // lstm
    bool graph_launch = false;

    /** Setup environments. */
    if (argc >= 7) {
        graph_launch = static_cast<bool>(atoi(argv[1]));
        batch_size = static_cast<uint64_t>(atoi(argv[2]));
        seq_length = static_cast<uint64_t>(atoi(argv[3]));
        input_size = static_cast<uint64_t>(atoi(argv[4]));
        hidden_size = static_cast<uint64_t>(atoi(argv[5]));
        num_layer = static_cast<uint64_t>(atoi(argv[6]));
    }
   
    /** Declare handle and status. */ 
    //hipError_t cuda_status;    
    hipblasStatus_t cublas_status;
    hipblasHandle_t cublas_handle;

    /** Declare cublas algorithms. */
    hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;

    /** Setup GPU property. */
    int gpuid = 0;
    checkCudaErrors(hipSetDevice(gpuid));

    /** Initialize the cublas handle. */
    checkCublasErrors(hipblasCreate(&cublas_handle));
   
    /** Setup math property. */
    typedef __half io_type;
    typedef float math_type;
    hipblasMath_t cublas_math_mode = HIPBLAS_TENSOR_OP_MATH;
    hipblasSetMathMode(cublas_handle, cublas_math_mode); 

    /** Setup algorithm. */
    if (argc >= 8) {
        algo = static_cast<hipblasGemmAlgo_t>(atoi(argv[7]));
    }

    /** Create sperate stream for layer gemm and recurrent gemm(high priority). */
    hipStream_t stream[num_layer];
    for (int i=0; i<num_layer; i++)  {
        checkCudaErrors(hipStreamCreateWithPriority(&stream[i],
            hipStreamNonBlocking, -1));
    }

    /** Create events to synchronize layer gemm and recurrent gemm. */
    hipEvent_t events[num_layer][seq_length];
    for (int i=0; i<num_layer; i++)  {
        for (int j=0; j<seq_length; j++)  {
            checkCudaErrors(hipEventCreate(&events[i][j]));
        }
    }

    /** Create events to monitor time elapsed. */
    float milliseconds = 0;
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    /** Setup gemm dimensions. */
    int m = num_mat * hidden_size;
    int k = input_size + hidden_size;
    int n = batch_size;

    /**
     * Allocate matrix used by layer gemm and recurrent gemm.
     * Just to see the perf, initialization doesn't matter.
     */
    float *mw[num_layer];
    float *mi[num_layer];
    float *mo[num_layer];
    for (int i=0; i<num_layer; i++)  {
        checkCudaErrors(hipMalloc((void **)&mw[i], sizeof(io_type) * m * k));
        checkCudaErrors(hipMalloc((void **)&mi[i], sizeof(io_type) * k * n));
        checkCudaErrors(hipMalloc((void **)&mo[i], sizeof(io_type) * m * n));
    }
    
    /** Cudnn 7.1 use nn gemm for fprop. */
    hipblasOperation_t transa = HIPBLAS_OP_N, transb = HIPBLAS_OP_N;

    /** Setup constant. */
    float alpha = 1.0f, beta = 0.0f;

    /** Graph launch. */
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    uint64_t num_iter;
    if (graph_launch)  {
        num_iter = 1;
        checkCudaErrors(hipGraphCreate(&graph, 0));
        checkCudaErrors(hipStreamBeginCapture(stream[0], hipStreamCaptureModeGlobal));
    } else {
        num_iter = ITERATIONS;
        checkCudaErrors(hipEventRecord(start));
    }

    /** Emulate the cudnn lstm fprop. */
    for (int iter=0; iter<num_iter; iter++) {
        for (int d=0; d<num_layer+seq_length-1; d++)  {
            for (int i=0; i<seq_length; i++)  {
                /** Boundary check. */
                int l = d - i;
                if (l < 0 || l >= num_layer)  {
                    continue;
                }

                /** Layer limitation. */
                if (l > 0)  {
                    checkCudaErrors(hipStreamWaitEvent(stream[l], events[l-1][i], 0));
                }

                /** Combined GEMM. */
                checkCublasErrors(hipblasSetStream(cublas_handle, stream[l]));
                cublas_status = hipblasGemmEx(cublas_handle,
                                transa, transb,
                                m, n, k,
                                &alpha,
                                mw[l], HIP_R_16F, transa == HIPBLAS_OP_N ? m : k,
                                mi[l], HIP_R_16F, k,
                                &beta,
                                mo[l], HIP_R_16F, m,
                                HIP_R_32F,
                                algo);
                checkCublasErrors(cublas_status);

                /** Element-wise operation. */
                checkCudaErrors(hipEventRecord(events[l][i], stream[l]));
            }

        }

        /**
         * Make sure all works done.
         * Stream capture requires same stream in, same stream out.
         */
        for (int i=0; i<num_layer; i++)  {
            checkCudaErrors(hipStreamWaitEvent(stream[0],
                events[i][seq_length-1], 0));
        }
    }

    if (graph_launch)  {
        checkCudaErrors(hipStreamEndCapture(stream[0], &graph));
        checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

        checkCudaErrors(hipEventRecord(start));
        for (int i = 0; i < ITERATIONS; i++)  {
            checkCudaErrors(hipGraphLaunch(graphExec, 0));
        }
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    //checkCudaErrors(hipStreamSynchronize(0));
    //checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    /** Print time and gflops. */
    double fma = ITERATIONS * num_layer * num_mat * hidden_size * (input_size + hidden_size) * batch_size * seq_length;
    std::printf("CUDA elapsed: %.3f ms, %.3f Tflops\n", \
        milliseconds/ITERATIONS, fma*2/milliseconds/1e9);
    
    /** Free the device memory. */
    for (int i=0; i<num_layer; i++)  {
        checkCudaErrors(hipFree(mw[i]));
        checkCudaErrors(hipFree(mi[i]));
        checkCudaErrors(hipFree(mo[i]));
    }

    /** Destroy events. */
    for (int i=0; i<num_layer; i++)  {
        for (int j=0; j<seq_length; j++)  {
            checkCudaErrors(hipEventDestroy(events[i][j]));
        }
    }
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    /** Destory streams. */
    for (int i=0; i<num_layer; i++)  {
        checkCudaErrors(hipStreamDestroy(stream[i]));
    }

    /** Destroy CUDA graph. */
    if (graph_launch) {
        checkCudaErrors(hipGraphDestroy(graph));
        checkCudaErrors(hipGraphExecDestroy(graphExec));
    }

    /** Free the cublas handle. */
    checkCublasErrors(hipblasDestroy(cublas_handle));

    return 0;
}
